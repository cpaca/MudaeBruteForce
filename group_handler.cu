#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "constants.cuh"
#include "host_device_helper.cuh"
#include "group_handler.cuh"
#include <iostream>
#include <cassert>

groupNum* host_groupData = nullptr;
groupNum* host_rowIndices = nullptr;
groupNum host_numRows = 0;
__device__ groupNum* dev_groupData = nullptr;
__device__ groupNum* dev_rowIndices = nullptr;
__device__ groupNum dev_numRows = 0;

// I understand that these functions should logically be at the bottom of the file, not near the top,
// but when I put it near the bottom, I can't compile.
// It's weird, but this is easier than finding a fix.
__global__ void groupDataDeviceValidate() {
	// I understand I could just do this in one for-loop
	// but this is a more "genuine" representation of what each row represents
	/*
	for (int i = 0; i < dev_numRows; i++) {
		groupNum rowStart = dev_rowIndices[i]; // inclusive
		groupNum rowEnd = dev_rowIndices[i + 1]; // exclusive

		printf("Row data: ");
		for (int j = rowStart; j < rowEnd; j++) {
			printf("%u ", dev_groupData[j]);
		}
		printf("\n");

		// ... visual studio is too dumb so only have nvcc process this part
#ifdef __CUDA_ARCH__
		// Without this the printf buffer gets too filled up.
		// Performance isn't really necessary here since we just do a visual check.
		__nanosleep(1000000);
#endif
	}
	printf("End of groupDataDeviceValidate\n");
	//*/
}

__global__ void getGroupDataDeviceValidate() {

}

__host__ __device__ groupType getGroupData(groupNum numGroup)
{
	// Like the comment said, 1-indexed.
	groupNum rowNum = numGroup;

	// Technically this variable renaming procedure isn't necessary
	// but it reduces a LOT of code reuse and also helps visual studio at least somewhat understand what's going on.
	groupNum* groupData;
	groupNum* rowIndices;
	groupNum numRows;

#ifdef __CUDA_ARCH__
	// Device side
	groupData = dev_groupData;
	rowIndices = dev_rowIndices;
	numRows = dev_numRows;
#else
	// Host side
	groupData = host_groupData;
	rowIndices = host_rowIndices;
	numRows = host_numRows;
#endif

	// exists in device cuda natively
	// and in host code with cassert
	assert(rowNum < numRows);

	groupNum dataIdx = rowIndices[rowNum];
	groupNum nextIdx = rowIndices[rowNum + 1];
	groupNum* dataPtr = groupData + dataIdx;

	groupType out;

	out.weight = dataPtr[0];
	out.value = dataPtr[1];
	out.numBundles = nextIdx - dataIdx;
	out.bundles = dataPtr + 2;

	return out;
}

__host__ void saveAllGroupData(groupNum* groupData, groupNum* rowIndices, groupNum numRows)
{
	// Array validation host-side:

	/*
	std::cout << "Reconstructing groupData..." << "\n";

	for (int i = 0; i < numRows; i++) {
		groupNum startIdx = rowIndices[i];
		groupNum endIdx = rowIndices[i + 1];
		for (int i = startIdx; i < endIdx; i++) {
			std::cout << groupData[i] << " ";
		}
		std::cout << "\n";
	}
	//*/

	host_groupData = groupData;
	host_rowIndices = rowIndices;
	host_numRows = numRows;

	// I understand that this is "sort of" a 2D array so theoretically, I should use hipMallocPitch, however:
	// - This is NOT a proper 2D array; the first row always has 2 elements and the last (many) rows always have at least 3 elements
	// --- (Also, some rows may have 4, 5, 6, etc. elements, so this is even less of a 2D array)
	// --- so hipMallocPitch probably wouldn't function properly here
	// Basically, as I understand it, hipMallocPitch is built for rectangular 2D arrays
	// This isn't rectangular.
	auto devClone_groupData = hostArrayToDevice(host_groupData, host_rowIndices[host_numRows]);
	auto devClone_rowIndices = hostArrayToDevice(host_rowIndices, host_numRows);

	hipMemcpyToSymbol(HIP_SYMBOL(dev_groupData), &devClone_groupData, sizeof(devClone_groupData));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_rowIndices), &devClone_rowIndices, sizeof(devClone_rowIndices));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_numRows), &host_numRows, sizeof(host_numRows));

	groupDataDeviceValidate<<<1, 1 >>>();
}