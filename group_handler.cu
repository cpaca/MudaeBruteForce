#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "constants.cuh"
#include "memory_handler.cuh"
#include <iostream>

groupType* host_groupData = nullptr;
groupType* host_rowIndices = nullptr;
groupType host_numRows = 0;
__device__ groupType* dev_groupData = nullptr;
__device__ groupType* dev_rowIndices = nullptr;
__device__ groupType dev_numRows = 0;

// I understand that this function should logically be at the bottom of the file, not near the top,
// but when I put it near the bottom, I can't compile.
// It's weird, but this is easier than finding a fix.
__global__ void groupDataDeviceValidate() {
	// I understand I could just do this in one for-loop
	// but this is a more "genuine" representation of what each row represents
	/*
	for (int i = 0; i < dev_numRows; i++) {
		groupType rowStart = dev_rowIndices[i]; // inclusive
		groupType rowEnd = dev_rowIndices[i + 1]; // exclusive

		printf("Row data: ");
		for (int j = rowStart; j < rowEnd; j++) {
			printf("%u ", dev_groupData[j]);
		}
		printf("\n");

		// ... visual studio is too dumb so only have nvcc process this part
#ifdef __CUDA_ARCH__
		// Without this the printf buffer gets too filled up.
		// Performance isn't really necessary here since we just do a visual check.
		__nanosleep(1000000);
#endif
	}
	printf("End of groupDataDeviceValidate\n");
	//*/
}

__host__ groupType* hostArrayToDevice(groupType* arr, int size)
{
	groupType* ret = (groupType*)cudaMallocSafe(size * sizeof(groupType));
	hipMemcpy(ret, arr, size * sizeof(groupType), hipMemcpyHostToDevice);
	return ret;
}

__host__ void saveGroupData(groupType* groupData, groupType* rowIndices, groupType numRows)
{
	// Array validation host-side:

	/*
	std::cout << "Reconstructing groupData..." << "\n";

	for (int i = 0; i < numRows; i++) {
		groupType startIdx = rowIndices[i];
		groupType endIdx = rowIndices[i + 1];
		for (int i = startIdx; i < endIdx; i++) {
			std::cout << groupData[i] << " ";
		}
		std::cout << "\n";
	}
	//*/

	host_groupData = groupData;
	host_rowIndices = rowIndices;
	host_numRows = numRows;

	// I understand that this is "sort of" a 2D array so theoretically, I should use hipMallocPitch, however:
	// - This is NOT a proper 2D array; the first row always has 2 elements and the last (many) rows always have at least 3 elements
	// --- (Also, some rows may have 4, 5, 6, etc. elements, so this is even less of a 2D array)
	// --- so hipMallocPitch probably wouldn't function properly here
	// Basically, as I understand it, hipMallocPitch is built for rectangular 2D arrays
	// This isn't rectangular.
	auto devClone_groupData = hostArrayToDevice(host_groupData, host_rowIndices[host_numRows]);
	auto devClone_rowIndices = hostArrayToDevice(host_rowIndices, host_numRows);

	hipMemcpyToSymbol(HIP_SYMBOL(dev_groupData), &devClone_groupData, sizeof(devClone_groupData));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_rowIndices), &devClone_rowIndices, sizeof(devClone_rowIndices));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_numRows), &host_numRows, sizeof(host_numRows));

	groupDataDeviceValidate<<<1, 1 >>>();
}