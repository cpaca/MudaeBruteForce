#include "hip/hip_runtime.h"
#include ""
#include "error_handler.cuh"
#include "constants.cuh"
#include <iostream>

// It's possible that these could be inlined safely
// However until it becomes an issue I don't think I'm gonna do that.
// The compiler will probably do it for me, but at least then it's the compiler being a million times smarter than I am.

__host__ void* cudaMallocSafe(size_t size) {
	void* dev_out;
	hipError_t err = hipMalloc(&dev_out, size);
	CUDAErrorCheck(err);
	// printf("Allocating memory of size %u\n", size);
	return dev_out;
}

__host__ void* cudaMallocManagedSafe(size_t size, unsigned int flags = hipMemAttachGlobal) {
	void* dev_out;
	hipError_t err = hipMallocManaged(&dev_out, size, flags);
	CUDAErrorCheck(err);
	return dev_out;
}

__host__ void cudaFreeSafe(void* devPtr) {
	hipError_t err = hipFree(devPtr);
	CUDAErrorCheck(err);
}

__host__ groupNum* hostArrayToDevice(groupNum* arr, int size)
{
	groupNum* ret = (groupNum*)cudaMallocSafe(size * sizeof(groupNum));
	hipMemcpy(ret, arr, size * sizeof(groupNum), hipMemcpyHostToDevice);
	return ret;
}