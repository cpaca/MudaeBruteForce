#include "hip/hip_runtime.h"
#define NUM_CLOCKS 4
#define PROFILING_STR_WIDTH 45

// If this is set to false, then no profiling is done
// and you can treat all of the functions in this file like they are empty.
// However, the function signatures are not removed so that you don't have to comment out the functions
// on other files.
#define PROFILE true

// Checkpoints and variables.

__device__ size_t numThreads = 0;

__device__ size_t getTaskCheckpoint = 0;
__device__ size_t validTaskCheckpoint = 0;
__device__ size_t copyTaskCheckpoint = 0;
__device__ size_t makeNewTaskCheckpoint = 0;
__device__ size_t fullTaskCheckpoint = 0;
__device__ size_t bundlePtrCheckpoint = 0;
__device__ size_t activateBundleCheckpoint = 0;
__device__ size_t deleteSetCheckpoint = 0;
__device__ size_t finishLoopCheckpoint = 0;

__device__ size_t tasksCreated = 0;
__device__ size_t tasksRezzed = 0;

// The actual computation functions and whatnot.

__device__ size_t* initProfiling(){
#if PROFILE
    atomicAdd(&numThreads, 1);

    auto* clocks = new size_t[NUM_CLOCKS];
    for(size_t i = 0; i < NUM_CLOCKS; i++){
        // effectively -1
        // but way easier to detect (or ignore?) problems with
        clocks[i] = ~0;
    }
    return clocks;
#endif
}

__device__ void destructProfiling(const size_t* clocks){
#if PROFILE
    delete[] clocks;
#endif
}

__device__ void startClock(size_t *clocks, int clockNum) {
#if PROFILE
    clocks[clockNum] = clock();
#endif
}

__device__ void checkpoint(size_t *clocks, int clockNum, size_t* saveTo) {
#if PROFILE
    size_t endTime = clock();
    size_t deltaTime = endTime - clocks[clockNum];
    // was getting some errors with profiling giving VERY wrong numbers
    // and it turns out doing this fixes it!
    // Note: 2.5 seconds of computation is less than 10B, so >2Trillion is a really shitty step taking a very long time
    if(deltaTime < 2000000000000L){
        atomicAdd(saveTo, deltaTime);
    }
    // don't reset the clock with clock64()
    // because atomicAdd can take a very, very long time in bad cases
    clocks[clockNum] = clock();
#endif
}

/**
 * Increments a certain value, if the profiler is running.
 * This is helpful for saying how many times certain things happened
 * but this is also a profiler thing so it self-disables when you stop profiling
 */
__device__ void profileIncrement(size_t* saveTo){
#if PROFILE
    atomicAdd(saveTo, 1);
#endif
}

__host__ std::string padStr(const std::string& str){
#if PROFILE
    int missingLength = PROFILING_STR_WIDTH - str.length();
    if(missingLength <= 0){
        return str;
    }
    return str + std::string(missingLength, ' ');
#else
    return "";
#endif
}

__host__ void printProfilingStrNum(const std::string& str, size_t &deviceSymbol, const size_t totalThreads = 0){
#if PROFILE
    size_t num;
    hipMemcpyFromSymbol(&num, HIP_SYMBOL(deviceSymbol), sizeof(size_t));
    if(totalThreads != 0) {
        num /= totalThreads;
    }
    std::cout << padStr(str) << std::to_string(num) << std::endl;
#endif
}

__host__ void printProfilingData(){
#if PROFILE
    size_t totalThreads;
    hipMemcpyFromSymbol(&totalThreads, HIP_SYMBOL(numThreads), sizeof(size_t));
    std::cout << "Threads counted: " << std::to_string(totalThreads) << "\n";
    printProfilingStrNum("Avg. time used getting the task: ", getTaskCheckpoint, totalThreads);
    printProfilingStrNum("Avg. time used validating the task: ", validTaskCheckpoint, totalThreads);
    printProfilingStrNum("Avg. time used copying new task: ", copyTaskCheckpoint, totalThreads);
    printProfilingStrNum("Avg. time used initializing new task: ", makeNewTaskCheckpoint, totalThreads);
    printProfilingStrNum("Avg. time used deleting an overfilled set: ", fullTaskCheckpoint, totalThreads);
    printProfilingStrNum("Avg. time used getting the bundle ptr: ", bundlePtrCheckpoint, totalThreads);
    printProfilingStrNum("Avg. time used deleting the bundle's series: ", activateBundleCheckpoint, totalThreads);
    printProfilingStrNum("Avg. time used finishing deleteSet: ", deleteSetCheckpoint, totalThreads);
    printProfilingStrNum("Avg. time used postprocessing: ", finishLoopCheckpoint, totalThreads);
    std::cout << "\n";
    printProfilingStrNum("Number of tasks created: ", tasksCreated);
    printProfilingStrNum("Number of tasks resurrected: ", tasksRezzed);
    std::cout << "\n";

    TaskQueue host_deadTaskQueue;
    hipMemcpyFromSymbol(&host_deadTaskQueue, HIP_SYMBOL(deadTaskQueue), sizeof(TaskQueue));
    std::cout << "Dead queue fullness: " << std::to_string(host_deadTaskQueue.writeIdx - host_deadTaskQueue.readIdx) << "\n";
    std::cout << "Dead queue writeIdx: " << std::to_string(host_deadTaskQueue.writeIdx) << "\n";


    std::cout << std::endl;
#endif
}