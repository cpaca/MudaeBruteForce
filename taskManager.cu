#include "hip/hip_runtime.h"
#include "task.cu"
#include <thrust/sort.h>
#define QUEUE_SIZE 20
#define QUEUE_ELEMENTS (((size_t) 1) << QUEUE_SIZE)

/**
 * Gets a task from the task queue.
 * If there are no tasks available, returns nullptr.
 * @return
 */
__device__ Task* getTask(TaskQueue &tasks){
    // TODO rewrite using atomicAdd
    //  - low priority, is just an optimization idea
    size_t offset = (threadIdx.x % 32) + 1;
    while(true){
        offset = min(offset, offset-1);
        size_t expectedReadIdx = tasks.readIdx + offset;
        if(expectedReadIdx >= tasks.writeIdx){
            return nullptr;
        }

        // Attempt to acquire this read idx...
        size_t atomicReadIdx = atomicCAS(&(tasks.readIdx), expectedReadIdx, expectedReadIdx+1);
        if(atomicReadIdx != expectedReadIdx){
            // Some other thread got the readIdx, so we gotta try again.
            continue;
        }

        char* queueAddress = (char*) tasks.queue;
        size_t queueIdx = expectedReadIdx % QUEUE_ELEMENTS;
        char* taskAddress = queueAddress + (queueIdx * queuePitch);
        Task* ret = (Task*) taskAddress;

        // TODO come up with a better fix?
        //  - very low priority, honestly it's just a CPU optimization
        //  and the hipMallocPitch will never go to less than 2 pages with a MAX_DL of 55ish
        // This is VERY VERY BAD programming practice to have repeated code in 3 places
        size_t taskStructBytes = sizeof(Task);
        size_t bundlesUsedBytes = sizeof(size_t) * setBundlesSetSize;
        size_t disabledSetsBytes = sizeof(size_t) * ((size_t) DISABLED_SETS_SIZE);
        size_t taskTotalBytes = taskStructBytes+bundlesUsedBytes+disabledSetsBytes;
        ret->bundlesUsed = (size_t*) (taskAddress + taskStructBytes);
        ret->disabledSets = (size_t*) (taskAddress + taskStructBytes + bundlesUsedBytes);

        return ret;
    }
}

/**
 * Puts a task into the task queue.
 */
__device__ void putTask(TaskQueue &tasks, Task* task){
    if(task == nullptr){
        return;
    }
    size_t putIdx = atomicAdd(&(tasks.writeIdx), 1);
    size_t queueIdx = putIdx % QUEUE_ELEMENTS;
    char* queueAddress = (char*) tasks.queue;
    char* taskAddress = queueAddress + (queueIdx * queuePitch);

    char* destAddress = taskAddress;
    char* srcAddress = (char*) task;
    memcpy(destAddress, srcAddress, queuePitch);
}

__host__ TaskQueue makeBlankTaskQueue() {
    // This is VERY VERY BAD programming practice to have repeated code in 3 places
    size_t taskStructBytes = sizeof(Task);
    size_t bundlesUsedBytes = sizeof(size_t) * host_setBundlesSetSize;
    size_t disabledSetsBytes = DISABLED_SETS_SIZE * sizeof(size_t);
    size_t taskTotalBytes = taskStructBytes+bundlesUsedBytes+disabledSetsBytes;

    TaskQueue ret;

    size_t host_queuePitch;
    cudaErrorCheck(
            hipMallocPitch(&ret.queue, &host_queuePitch, taskTotalBytes, QUEUE_ELEMENTS),
            "makeBlankTaskQueue mallocPitch error");
    hipMemcpyToSymbol(HIP_SYMBOL(queuePitch), &host_queuePitch, sizeof(host_queuePitch));

    ret.readIdx = 0;
    ret.writeIdx = 0;
    return ret;
}

/**
 * Copies the outTaskQueue to the inTaskQueue
 */
__host__ void reloadTaskQueue(bool incrementSDI = true){
    knapsackReload();

    // Get and swap the two task queues
    // Note that device inTaskQueue goes to host outTaskQueue
    TaskQueue host_inTaskQueue;
    TaskQueue host_outTaskQueue;
    hipMemcpyFromSymbol(&host_outTaskQueue, HIP_SYMBOL(inTaskQueue), sizeof(TaskQueue));
    hipMemcpyFromSymbol(&host_inTaskQueue, HIP_SYMBOL(outTaskQueue), sizeof(TaskQueue));

    // Save data for debug...
    size_t numTasks = host_inTaskQueue.writeIdx - host_inTaskQueue.readIdx;

    // Reset...
    host_outTaskQueue.readIdx = 0;
    host_outTaskQueue.writeIdx = 0;

    // Reload...
    hipMemcpyToSymbol(HIP_SYMBOL(inTaskQueue), &host_inTaskQueue, sizeof(TaskQueue));
    hipMemcpyToSymbol(HIP_SYMBOL(outTaskQueue), &host_outTaskQueue, sizeof(TaskQueue));

    // Update the expected setDeleteIndex
    if(incrementSDI){
        setDeleteIndex++;
    }
    size_t host_expectedSetToDelete = host_setDeleteOrder[setDeleteIndex];
    hipMemcpyToSymbol(HIP_SYMBOL(expectedSetToDelete), &host_expectedSetToDelete, sizeof(size_t));

    // Print some stuff for debug reasons
    std::cout << "With a setDeleteIndex of " << std::to_string(setDeleteIndex) << ",\n";
    std::cout << "the inTaskQueue has " << std::to_string(numTasks) << " tasks\n" << std::endl;
}

// The kernel-side function that assists with initializing the taskQueue
__global__ void kernelInitTaskQueue(size_t numSeries, size_t numBundles){
    // This is VERY VERY BAD programming practice to have repeated code in 3 places
    size_t taskStructBytes = sizeof(Task);
    size_t bundlesUsedBytes = sizeof(size_t) * setBundlesSetSize;
    size_t disabledSetsBytes = sizeof(size_t) * ((size_t) DISABLED_SETS_SIZE);
    size_t taskTotalBytes = taskStructBytes+bundlesUsedBytes+disabledSetsBytes;

    // Create and init task:
    // Init malloc-related stuff
    char* baseAddress = (char*) malloc(queuePitch);
    Task* taskAddress = (Task*) (baseAddress);
    auto* bundlesUsedAddress = (size_t*) (baseAddress + taskStructBytes);
    memset(bundlesUsedAddress, 0, bundlesUsedBytes);
    auto* disabledSetsAddress = (size_t*) (baseAddress + taskStructBytes + bundlesUsedBytes);

    taskAddress->bundlesUsed = bundlesUsedAddress;
    taskAddress->disabledSets = disabledSetsAddress;

    // Init simple constants
    taskAddress->remainingOverlap = OVERLAP_LIMIT;
    taskAddress->DLSlotsRemn = MAX_DL;
    taskAddress->disabledSetsIndex = 0;
    taskAddress->score = 0;

    // Init complex constants
    // Proper init score and disabledSetsIndex... and disabledSets
    auto** bundlePtrs = new size_t*[numBundles];
    for(size_t i = 0; i < numBundles; i++){
        if(freeBundles[i] != 0){
            size_t setNum = numSeries + i;
            activateBundle(numSeries, taskAddress, setNum);
            taskAddress->disabledSets[taskAddress->disabledSetsIndex] = setNum;
            bundlePtrs[taskAddress->disabledSetsIndex] = bundleSeries + bundleIndices[i];
            taskAddress->disabledSetsIndex++;
        }
    }

    for(size_t seriesNum = 0; seriesNum < numSeries; seriesNum++){
        bool addSeries = false;
        for(size_t i = 0; i < taskAddress->disabledSetsIndex; i++){
            size_t bundleSetNum = *(bundlePtrs[i]);
            while(bundleSetNum < seriesNum){
                bundlePtrs[i]++;
                bundleSetNum = *(bundlePtrs[i]);
            }
            if(bundleSetNum == seriesNum){
                addSeries = true;
            }
        }

        if(addSeries){
            taskAddress->score += deviceSeries[(2 * seriesNum) + 1];
        }
    }
    delete[] bundlePtrs;

    putTask(outTaskQueue, taskAddress);

    auto* queue = (std::uint8_t*) outTaskQueue.queue;
    Task* task = (Task*) queue;

    free(baseAddress);
}

__host__ void initTaskQueue(size_t numSeries, size_t numBundles){
    // This is a weird way to do it, but doing it this way lets me basically 1:1 repeat other code.
    TaskQueue host_inTaskQueue = makeBlankTaskQueue();
    hipMemcpyToSymbol(HIP_SYMBOL(inTaskQueue), &host_inTaskQueue, sizeof(TaskQueue));

    TaskQueue host_outTaskQueue = makeBlankTaskQueue();
    hipMemcpyToSymbol(HIP_SYMBOL(outTaskQueue), &host_outTaskQueue, sizeof(TaskQueue));

    cudaErrorCheck(hipDeviceSynchronize(), "initTaskQueue first synchronize invoked a CUDA error");

    kernelInitTaskQueue<<<1, 1>>>(numSeries, numBundles);
    cudaErrorCheck(hipDeviceSynchronize(), "initTaskQueue second synchronize invoked a CUDA error");

    // Since kernelInitTaskQueue calls putTask and I plan to make putTask go to outQueue only
    // this is how I have to swap the input and output sides.
    std::cout << "InitTaskQueue calling Reload\n";
    reloadTaskQueue(false);
    std::cout << "InitTaskQueue done calling Reload\n";
}