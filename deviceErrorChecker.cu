#include <cassert>

__host__ void cudaErrorCheck(hipError_t error, const std::string& str) {
    if (error != hipSuccess) {
        std::cout << "Caught a CUDA error. Message: " << str << "\n";
        std::cout << "Error type: " << hipGetErrorName(error) << "\n";
        std::cout << "Error string: " << hipGetErrorString(error) << std::endl;

        assert(false);
    }
}