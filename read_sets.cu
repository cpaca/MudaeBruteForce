#include "hip/hip_runtime.h"
#include <fstream>
#include <thrust/host_vector.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "constants.cuh"
#include "vector_helper.cuh"
#include "memory_handler.cuh"
#include "group_handler.cuh"

// Note to self: Use thrust
// https://docs.nvidia.com/cuda/thrust/index.html

__host__ std::string getToken(std::string& str) {
	auto idx = str.find("$");

	std::string token;
	if (idx == std::string::npos) {
		// didn't find delimiter
		// therefore no more delimiters
		token = str;
		str = "";
	}
	else {
		token = str.substr(0, idx);
		str = str.substr(idx + 1);
	}
	return token;
}

// I understand that this function should logically be at the bottom of the file, not near the top,
// but when I put it near the bottom, I can't compile.
// It's weird, but this is easier than finding a fix.
__global__ void readFileDeviceValidate() {
	// I understand I could just do this in one for-loop
	// but this is a more "genuine" representation of what each row represents
	/*
	for (int i = 0; i < dev_numRows; i++) {
		groupType rowStart = dev_rowIndices[i]; // inclusive
		groupType rowEnd = dev_rowIndices[i + 1]; // exclusive

		printf("Row data: ");
		for (int j = rowStart; j < rowEnd; j++) {
			printf("%u ", dev_groupData[j]);
		}
		printf("\n");
	}
	*/
}

__host__ groupType* hostArrayToDevice(groupType* arr, int size)
{
	groupType* ret = (groupType*) cudaMallocSafe(size * sizeof(groupType));
	hipMemcpy(ret, arr, size * sizeof(groupType), hipMemcpyHostToDevice);
	return ret;
}

__host__ void readFile() {
	// Probably not the best method, but it's the best way to avoid exposing the structure of my computer
	// and also makes it so this wont break if i re-build this on a different computer... assuming i have working-data.
	std::string fileName = std::string(__FILE__) + "\\..\\working-data\\series.txt";

	std::ifstream file;
	file.open(fileName);

	std::string line;

	// We can't have a non-rectangular 2D array in device memory
	// but we can have a very long 1D array and then make a note of where each row starts and stops.
	// so if we have [[1,2], [3,4,5,6,7], [8], [9,10]], we would make note of the 1, 3, 8, and 9.
	thrust::host_vector<groupType> groupData;
	thrust::host_vector<groupType> rowIndices;

	while (std::getline(file, line)) {
		// Validation that readline works
		// std::cout << line << "\n";
		
		// Start of a new row.
		auto size = groupData.size();
		rowIndices.push_back(size);

		while (!line.empty()) {
			auto token = getToken(line);

			groupType num = (groupType) stoi(token);
			groupData.push_back(num);
		}
	}
	// Add one more so the very-last row knows where it ends.
	auto size = groupData.size();
	// If any of the elements of rowIndices exceeds 2billion, then this one will also exceed 2billion.
	// The only exception would be if there's over 9 quintillion elements in groupData, but for that to happen without an out-of-memory exception
	// would require approximately 36 exabytes of RAM.
	if (size > 2000000000) {
		// over 2 billion... might overload groupType.
		// I'm pretty sure this will never happen, but better to have the check.
		std::cerr << "SetType is going to get overloaded. Make it a bigger type.";
		exit(1);
	}
	rowIndices.push_back(size);

	// These vectors will never, ever be modified again (may be read again)
	// So shrink them to min size...
	groupData.shrink_to_fit();
	rowIndices.shrink_to_fit();

	auto arr_groupData = vectorToArray(groupData);
	auto arr_rowIndices = vectorToArray(rowIndices);
	auto numRows = rowIndices.size() - 1; // Note the last item represents the *end* of the last row
	saveGroupData(arr_groupData, arr_rowIndices, numRows);

	// I understand that this is "sort of" a 2D array so theoretically, I should use hipMallocPitch, however:
	// - This is NOT a proper 2D array; the first row always has 2 elements and the last (many) rows always have at least 3 elements
	// --- (Also, some rows may have 4, 5, 6, etc. elements, so this is even less of a 2D array)
	// --- so hipMallocPitch probably wouldn't function properly here
	// Basically, as I understand it, hipMallocPitch is built for rectangular 2D arrays
	// This isn't rectangular.
	// auto devClone_groupData = hostArrayToDevice(host_groupData, host_rowIndices[host_numRows]);
	// auto devClone_rowIndices = hostArrayToDevice(host_rowIndices, host_numRows);

	// hipMemcpyToSymbol(HIP_SYMBOL(dev_groupData), &devClone_groupData, sizeof(devClone_groupData));
	// hipMemcpyToSymbol(HIP_SYMBOL(dev_rowIndices), &devClone_rowIndices, sizeof(devClone_rowIndices));
	// hipMemcpyToSymbol(HIP_SYMBOL(dev_numRows), &host_numRows, sizeof(host_numRows));

	// readFileDeviceValidate<<<1, 1 >>>();
}