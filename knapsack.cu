#include "hip/hip_runtime.h"
__constant__ char* bestScores;
__device__ char* newBestScores;

// both of these variables are basically the width of each row in bytes
// just host vs device differences
// note that both are only written to ONE TIME (in knapsackInit) and never written to again
__constant__ size_t bestScoresPitch;
size_t host_pitch;
const size_t numRows = MAX_DL + 1;


__host__ void knapsackInit(){
    // named fake row width since host_pitch is the REAL row width
    size_t fakeRowWidth = (OVERLAP_LIMIT + 1) * sizeof(size_t);

    char* host_newBestScores;
    hipMallocPitch(&host_newBestScores, &host_pitch, fakeRowWidth, numRows);
    hipMemset(host_newBestScores, 0, host_pitch * numRows);
    hipMemcpyToSymbol(HIP_SYMBOL(bestScoresPitch), &host_pitch, sizeof(size_t));
    hipMemcpyToSymbol(HIP_SYMBOL(newBestScores), &host_newBestScores, sizeof(char*));

    char* host_bestScores;
    hipMalloc(&host_bestScores, host_pitch * numRows);
    hipMemset(host_bestScores, 0, host_pitch * numRows);
    hipMemcpyToSymbol(HIP_SYMBOL(bestScores), &host_bestScores, sizeof(char*));
}

__host__ void knapsackReload(){
    hipMemcpy(bestScores, newBestScores, host_pitch * numRows, hipMemcpyDeviceToDevice);
}

__device__ size_t knapsackReadBestScore(size_t rowNum, size_t colNum){
    char* rowAddress = bestScores + (rowNum * bestScoresPitch);
    auto* row = (size_t*) rowAddress;
    size_t* col = row + colNum;

    return *col;
}

__device__ bool knapsackIsTaskGood(Task* task){
    size_t DLSlotsRemn = task->DLSlotsRemn;
    size_t DLSlotsUsed = MAX_DL - DLSlotsRemn;

    size_t remainingOverlap = task->remainingOverlap;
    size_t overlapUsed = OVERLAP_LIMIT - remainingOverlap;

    size_t oldBestScore = knapsackReadBestScore(DLSlotsUsed, overlapUsed);

    return true;
}

__device__ void knapsackWrite(const size_t &DLSlotsUsed, const size_t &overlapUsed, const size_t &score){
    size_t rowNum = DLSlotsUsed;
    size_t colNum = overlapUsed;

    char* rowAddress = newBestScores + (rowNum * bestScoresPitch);
    auto* row = (size_t*) rowAddress;
    size_t* col = row + colNum;

    const size_t oldScore = atomicMax(col, score);
    // TODO propagate down the newBestScores knapsack
}

__device__ void knapsackWriteTask(Task* task){
    if(task == nullptr){
        return;
    }

    size_t DLSlotsRemn = task->DLSlotsRemn;
    size_t DLSlotsUsed = MAX_DL - DLSlotsRemn;

    size_t remainingOverlap = task->remainingOverlap;
    size_t overlapUsed = OVERLAP_LIMIT - remainingOverlap;

    knapsackWrite(DLSlotsUsed, overlapUsed, task->score);
}

#undef byte