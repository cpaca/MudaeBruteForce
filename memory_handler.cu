#include "hip/hip_runtime.h"
#include ""
#include "error_handler.cu"

// It's possible that these could be inlined safely
// However until it becomes an issue I don't think I'm gonna do that.
// The compiler will probably do it for me, but at least then it's the compiler being a million times smarter than I am.

/// <summary>
/// Similar to hipMalloc, however the error-handling is automatically done.
/// This means that the signature can be treated more like ordinary malloc.
/// </summary>
/// <param name="size">Number of bytes to allocate.</param>
/// <returns></returns>
__host__ void* cudaMallocSafe(size_t size) {
	void* dev_out;
	hipError_t err = hipMalloc(&dev_out, size);
	CUDAErrorCheck(err);
	return dev_out;
}

/// <summary>
/// Similar to hipMallocManaged, however the error-handling is automatically done.
/// This means that the signature can be treated more like ordinary malloc.
/// <para/> The "flags" parameter is still available if I ever need it, but looking at documentation, I highly doubt I will.
/// </summary>
/// <param name="size">Number of bytes to allocate.</param>
/// <returns></returns>
__host__ void* cudaMallocManagedSafe(size_t size, unsigned int flags = hipMemAttachGlobal) {
	void* dev_out;
	hipError_t err = hipMallocManaged(&dev_out, size, flags);
	CUDAErrorCheck(err);
	return dev_out;
}

/// <summary>
/// Similar to hipFree, however the error-handling is automatically done.
/// This means that the signature can be treated more like ordinary malloc.
/// </summary>
/// <param name="size"></param>
/// <returns></returns>
__host__ void cudaFreeSafe(void* devPtr) {
	hipError_t err = hipFree(devPtr);
	CUDAErrorCheck(err);
}