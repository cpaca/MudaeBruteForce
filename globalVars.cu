#include "hip/hip_runtime.h"
#ifndef MUDAEBRUTEFORCE_GLOBALVARS
#define MUDAEBRUTEFORCE_GLOBALVARS
#include "types.cu"
// Score to beat.
__device__ size_t bestScore = 0;

// For each bundle, what series are in it?
// (Index 0 is also the bundle's size.)
__device__ size_t* bundleSeries = nullptr;
// Index of each bundle in bundleSeries. So bundleSeries[bundleIndices[n]] is the first index of a bundle in bundleSeries.
__device__ size_t* bundleIndices = nullptr;

// For each set, what bundles contain it?
// The format is... kind of a long description.
// First, let setBundlesSetSize = (numBundles/sizeof(size_t))
// And for shorthand, let sBSS = setBundlesSetSize
// Indices setBundles[setNum * sBSS] to setBundles[(setNum+1)*sBSS - 1] are the indices for set setNum
// In other words, to loop over all values in setBundles relevant to a set:
// for(int i = 0; i < sBSS; i++){/*do something with setBundles[setNum*sBSS + i]*/}
//
// Now express setBundles[0], setBundles[1], ... as a bitstream.
// The first bit represents if the set is in bundle # 0
// The second bit represents if the set is in bundle # 1
// etc.
// Because this is a bitstream and size_t is 64-bits:
// the 65th bit (aka, the first bit of setBundles[1], aka setBundles[1]&0) represents if the set is in bundle #65
//
// Note that this is setBundles, so it needs to work for all SETS. Even Bundles.
// Also note that for bundles, their "bitstream" is all 0s except for itself, where it is 1.
__device__ size_t* setBundles = nullptr;
__constant__ size_t setBundlesSetSize = -1; // note that setBundles[-1] = illegal (unsigned type)

// Data about each series.
// deviceSeries[2n] is the size of series n
// deviceSeries[2n+1] is the value of series n
__device__ size_t* deviceSeries = nullptr;

// Free bundles.
// If freeBundles[n] is non-zero, then bundle n is free.
__device__ size_t* freeBundles = nullptr;

// The size of each set.
// Note that this is setSize_t, not size_t.
// This is important because of byte limitations.
__device__ setSize_t* global_setSizes = nullptr;
extern __shared__ setSize_t setSizes[];

// The order to delete each set in.
__device__ size_t* setDeleteOrder = nullptr;

// Turns out if I do this and #include this file, it works fine.
// Maximum number of bundles/series that can be activated.
const std::uint32_t MAX_DL = 50;
// Maximum number of free bundles.
// Can be changed whenever, but keep it low or CUDA will demand much more memory than necessary.
const std::uint32_t MAX_FREE_BUNDLES = 5;
// Overlap limit, defined in Mudae
const std::uint32_t OVERLAP_LIMIT = 30000;
// How many blocks to run.
// Note that each block gets 512 threads.
#define NUM_BLOCKS (1 << 12)
// "MinSize" is a variable determining the minimum size a series needs to be to be added to the DL.
// MinSize gets divided by 2 while the remainingOverlap exceeds minSize, so even a minSize of 2^31 will get fixed
// down to remainingOverlap levels.
// MAX_MINSIZE determines the maximum value minSize can be.
const std::uint32_t MAX_MINSIZE = 100;

#endif