#include "hip/hip_runtime.h"
#include <iostream>

__host__ void CUDAErrorCheck(hipError_t error) {
	if (error == hipSuccess) {
		// No error.
		return;
	}

	// Error detected!
	// Can __FILE__ access the parent file?
	// probably not tbh, it's a compile-time thing
	// Well, if it becomes a problem I investigate __FILE__ harder
	std::cerr << "CUDA Error detected: " << hipGetErrorString(error);
	exit(error);
}